
#include <hip/hip_runtime.h>
__device__ float bar(float a, float b) {
    return a + b;
}

__device__ void incrval(float *a) {
    *a += 3;
}

__global__ void somekernel1(float *a) {
    a[0] = a[1];
}

__global__ void foo(float *data) {
        data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}

template< typename T >
__device__ T addNumbers(T one, T two) {
    return one + two;
}

__global__ void use_template1(float *data, int *intdata) {
    data[0] = addNumbers(data[1], data[2]);
    intdata[0] = addNumbers(intdata[1], intdata[2]);
}

__global__ void someops_float(float *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] * data[2];
    // data[0] += log(data[1]);
    // data[0] += exp(data[1]);
    // data[0] += tanh(data[1]);
    data[0] -= sqrt(data[1]);
}

__global__ void someops_int(int *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] + data[2];
    data[0] += data[1] * data[2];
}
