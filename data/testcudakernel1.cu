
#include <hip/hip_runtime.h>
__device__ float bar(float a, float b) {
    return a + b;
}

__device__ void incrval(float *a) {
    *a += 3;
}

__global__ void somekernel1(float *a) {
    a[0] = a[1];
}

__global__ void foo(float *data) {
        data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}
