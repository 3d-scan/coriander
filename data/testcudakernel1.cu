
#include <hip/hip_runtime.h>
__device__ float bar(float a, float b) {
    return a + b;
}

__device__ void incrval(float *a) {
    *a += 3;
}

__global__ void somekernel1(float *a) {
    a[0] = a[1];
}

__global__ void foo(float *data) {
        data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}

template< typename T >
__device__ T addNumbers(T one, T two) {
    return one + two;
}

__global__ void use_template1(float *data, int *intdata) {
    data[0] = addNumbers(data[1], data[2]);
    intdata[0] = addNumbers(intdata[1], intdata[2]);
}
