#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__device__ float bar(float a, float b) {
    return a + b;
}

__device__ void incrval(float *a) {
    *a += 3;
}

__global__ void foo(float *data) {
    data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}

__global__ void use_tid2(int *data) {
    int tid = threadIdx.x;
    data[tid] = data[tid] + tid;
}

__global__ void copy_float(float *a) {
    a[0] = a[1];
}

__global__ void use_blockid(float *data) {
    int blkid = blockIdx.x;
    data[blkid] = 123.0f;
}

__global__ void use_griddim(float *data) {
    int blkid = gridDim.x;
    data[blkid] = 123.0f;
}

__global__ void use_blockdim(float *data) {
    int blkid = blockDim.x;
    data[blkid] = 123.0f;
}

template< typename T >
__device__ T addNumbers(T one, T two) {
    return one + two;
}

__global__ void use_template1(float *data, int *intdata) {
    if(threadIdx.x == 0) {
        data[0] = addNumbers(data[1], data[2]);
        intdata[0] = addNumbers(intdata[1], intdata[2]);
    }
}

__host__ float someHostFunction(float input) {
    cout << "You called: someHostFunction()" << endl;
    return input * 100.0f;
}

__global__ void someops_float(float *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] * data[2];
    data[0] += log(data[1]);
    data[0] += exp(data[1]);
    data[0] += tanh(data[1]);
    data[0] -= sqrt(data[1]);
}

__global__ void someops_int(int *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] + data[2];
    data[0] += data[1] * data[2];
    data[0] += data[1] << data[2];
    data[0] += data[1] >> data[2];
}

__global__ void testbooleanops(int *data) {
    bool a = data[0] > 0;
    bool b = data[1] < 0;
    data[2] = (int)(a && b);
    data[3] = (int)(a || b);
    data[4] = (int)(!a);
}

__global__ void testcomparisons_int_signed(int *data) {
    data[5] = (int)(data[0] >= data[1]);
    data[6] = (int)(data[0] <= data[1]);
    data[7] = (int)(data[0] > data[1]);
    data[8] = (int)(data[0] < data[1]);
    data[9] = (int)(data[0] == data[1]);
    data[10] = (int)(data[0] != data[1]);
}

__global__ void testcomparisons_float(float *data) {
    data[5] = (data[0] >= data[1]);
    data[6] = (data[0] <= data[1]);
    data[7] = (data[0] > data[1]);
    data[8] = (data[0] < data[1]);
    data[9] = (data[0] == data[1]);
    data[10] = (data[0] != data[1]);
}

// // __global__ void testcomparisons_int_unsigned(unsigned int *data) {
// //     data[5] = (unsigned int)(data[0] >= data[1]);
// //     data[6] = (unsigned int)(data[0] <= data[1]);
// //     data[7] = (unsigned int)(data[0] > data[1]);
// //     data[8] = (unsigned int)(data[0] < data[1]);
// //     data[9] = (unsigned int)(data[0] == data[1]);
// //     data[10] = (unsigned int)(data[0] != data[1]);
// // }

__global__ void testsyncthreads(float *data) {
    int tid = threadIdx.x;
    data[tid] *= 2;
    syncthreads();
    data[tid + 1] += 2;
}

__global__ void testDoWhile(int *data, int N) {
    int p = threadIdx.x;
    do {
        p++;
    } while(data[p] != 0);
}

__global__ void testWhile(int *data, int N) {
    int p = threadIdx.x;
    while(data[p] != 0) {
        p++;
    }
}

__global__ void testIf(int *data, int N) {
    int tid = threadIdx.x;
    if(tid < N) {
        data[tid] *= 2;
    }
}

__global__ void testIfElse(int *data, int N) {
    int tid = threadIdx.x;
    if(tid < N) {
        data[tid] *= 2;
        data[tid + 3] *= 2;
    } else {
        data[tid] -= 20;
        data[tid + 5] -= 20;
    }
}

__global__ void testTernary(float *data) {
    data[0] = data[1] > 0 ? data[2] : data[3];
}

__global__ void testFor(float *data, int N) {
    if(threadIdx.x == 0) {
        float sum = 0.0f;
        for(int i = 0; i < N; i++) {
            sum += data[i];
        }
        data[0] = sum;
    }
}

void myprintint(int value) {
    cout << "myprintint " << value << endl;
}

void myprintfloat(float value) {
    cout << "myprintfloat " << value << endl;
}

void myprintvoidstar(void *value) {
    cout << "myprintvoid* " << value << endl;
}

void mynop() {

}

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

__host__ void launchSetValue(float *data, int idx, float value) {
    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(data, idx, value);
}

struct MyStruct {
    int x;
    float y;
};

__global__ void testStructs(MyStruct *structs, float *float_data, int *int_data) {
    int_data[0] = structs[0].x;
    float_data[0] = structs[0].y;
    float_data[1] = structs[1].y;
}

__global__ void testFloat4(float4 *data) {
    float4 myregister4 = data[1];
    float *myregisterfloat = (float *)&myregister4;
    myregisterfloat[1] = myregisterfloat[2] * myregisterfloat[3];
    data[0] = myregister4;
}

// __global__ void testFloat4_test2(float4 *data) {
//     data[0] = data[1];
// }

// __global__ void testFloat4_test3(float4 *data) {
//     float4 privateFloats[32];
//     for(int i = 0; i < 32; i++) {
//         privateFloats[i] = data[i];
//     }
//     for(int i = 0; i < 32; i++) {
//         data[i + 1] = privateFloats[i];
//     }
// }

__global__ void testLocal(float *data) {
    __shared__ float myshared[32];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
}

__global__ void testLocal2(float *data) {
    __shared__ float myshared[64];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
    myshared[tid + 1] = data[tid];
    data[1] = myshared[tid];
}

__global__ void testArray(float *data) {
    float privateFloats[32];
    for(int i = 0; i < 32; i++) {
        privateFloats[i] = data[i * 3];
    }
    for(int i = 0; i < 32; i+= 2) {
        data[i + 1] = privateFloats[i];
    }
}

__global__ void testmemcpy(float *data) {
    float privateFloats[32];
    for(int i = 0; i < 32; i++) {
        privateFloats[i] = data[i];
    }
    for(int i = 0; i < 32; i+= 2) {
        data[i] = privateFloats[i];
    }
}
