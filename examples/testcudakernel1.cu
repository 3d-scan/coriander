
#include <hip/hip_runtime.h>
__device__ float bar(float a, float b) {
    return a + b;
}

// __device__ void incrval(float *a) {
//     *a += 3;
// }

__global__ void foo(float *data) {
    data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}

__global__ void use_tid2(int *data) {
    int tid = threadIdx.x;
    data[tid] = data[tid] + tid;
}

__global__ void copy_float(float *a) {
    a[0] = a[1];
}

__global__ void use_blockid(float *data) {
    int blkid = blockIdx.x;
    data[blkid] = 123.0f;
}

__global__ void use_griddim(float *data) {
    int blkid = gridDim.x;
    data[blkid] = 123.0f;
}

__global__ void use_blockdim(float *data) {
    int blkid = blockDim.x;
    data[blkid] = 123.0f;
}

// template< typename T >
// __device__ T addNumbers(T one, T two) {
//     return one + two;
// }

// __global__ void use_template1(float *data, int *intdata) {
//     data[0] = addNumbers(data[1], data[2]);
//     intdata[0] = addNumbers(intdata[1], intdata[2]);
// }

// __host__ float someHostFunction(float input) {
//     return input * 100.0f;
// }

// __global__ void someops_float(float *data) {
//     data[0] = data[1] - data[2];
//     data[0] += data[1] / data[2];
//     data[0] += data[1] * data[2];
//     data[0] += log(data[1]);
//     data[0] += exp(data[1]);
//     data[0] += tanh(data[1]);
//     data[0] -= sqrt(data[1]);
// }

// __global__ void someops_int(int *data) {
//     data[0] = data[1] - data[2];
//     data[0] += data[1] / data[2];
//     data[0] += data[1] + data[2];
//     data[0] += data[1] * data[2];
//     data[0] += data[1] << data[2];
//     data[0] += data[1] >> data[2];
// }

// __global__ void testbooleanops(int *data) {
//     bool a = data[0] > 0;
//     bool b = data[1] < 0;
//     data[2] = (int)(a && b);
//     data[3] = (int)(a || b);
//     data[4] = (int)(!a);
// }

// __global__ void testcomparisons_int_signed(int *data) {
//     data[5] = (int)(data[0] >= data[1]);
//     data[6] = (int)(data[0] <= data[1]);
//     data[7] = (int)(data[0] > data[1]);
//     data[8] = (int)(data[0] < data[1]);
//     data[9] = (int)(data[0] == data[1]);
//     data[10] = (int)(data[0] != data[1]);
// }

// __global__ void testcomparisons_float(float *data) {
//     data[5] = (data[0] >= data[1]);
//     data[6] = (data[0] <= data[1]);
//     data[7] = (data[0] > data[1]);
//     data[8] = (data[0] < data[1]);
//     data[9] = (data[0] == data[1]);
//     data[10] = (data[0] != data[1]);
// }

// // __global__ void testcomparisons_int_unsigned(unsigned int *data) {
// //     data[5] = (unsigned int)(data[0] >= data[1]);
// //     data[6] = (unsigned int)(data[0] <= data[1]);
// //     data[7] = (unsigned int)(data[0] > data[1]);
// //     data[8] = (unsigned int)(data[0] < data[1]);
// //     data[9] = (unsigned int)(data[0] == data[1]);
// //     data[10] = (unsigned int)(data[0] != data[1]);
// // }

// __global__ void testsyncthreads(float *data) {
//     int tid = threadIdx.x;
//     data[tid] *= 2;
//     syncthreads();
//     data[tid + 1] += 2;
// }

// __global__ void testDoWhile(int *data, int N) {
//     int p = threadIdx.x;
//     do {
//         p++;
//     } while(data[p] != 0);
// }

// __global__ void testWhile(int *data, int N) {
//     int p = threadIdx.x;
//     while(data[p] != 0) {
//         p++;
//     }
// }

// __global__ void testIf(int *data, int N) {
//     int tid = threadIdx.x;
//     if(tid < N) {
//         data[tid] *= 2;
//     }
// }

// __global__ void testIfElse(int *data, int N) {
//     int tid = threadIdx.x;
//     if(tid < N) {
//         data[tid] *= 2;
//         data[tid + 3] *= 2;
//     } else {
//         data[tid] -= 20;
//         data[tid + 5] -= 20;
//     }
// }

// __global__ void testFor(float *data, int N) {
//     float sum = 0.0f;
//     for(int i = 0; i < N; i++) {
//         sum += data[i];
//     }
//     data[0] = sum;
// }

// __global__ void testTernary(float *data) {
//     data[0] = data[1] > 0 ? data[2] : data[3];
// }

// // __global__ void testFloat4(float4 *data) {
// //     float4 myregister4 = data[1];
// //     float *myregisterfloat = (float *)&myregister4;
// //     myregisterfloat[1] = myregisterfloat[2] * myregisterfloat[3];
// //     data[0] = myregister4;
// // }

// // __global__ void testLocal(float *data) {
// //     __shared__ float myshared[32];
// //     int tid = threadIdx.x;
// //     myshared[tid] = data[tid];
// //     data[0] = myshared[tid + 1];
// // }
