#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct MyStruct {
    float *floats;
    float afloat;
};

__global__ void getValue(struct MyStruct mystruct, float *data) {
    data[0] = mystruct.floats[0] + 3.0f;
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMalloc(&deviceFloats2, N * sizeof(float));

    MyStruct mystruct;
    mystruct.floats = (float *)deviceFloats1;
    hostFloats1[0] = 123;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, ((float *)deviceFloats2) + 0);

    // now copy back entire buffer
    // hostFloats[64] = 0.0f;
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats2[0] << endl;

    assert(hostFloats2[0] == 126);

    hipHostFree(hostFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats1);
    hipFree(deviceFloats2);
    hipStreamDestroy(stream);

    return 0;
}
