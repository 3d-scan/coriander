#include "hip/hip_runtime.h"
#include "test_callinternal.h"

#include <iostream>

__global__ void mykernel(MyStruct mystruct, float *data) {
    somefunc(mystruct);
    data[0] = mystruct.somefloats[0] * 2.0f;
}

int main(int argc, char *argv[]) {

    int N = 32;
    float *f1_host = new float[N];
    // MyStruct *struct_host = new MyStruct();
    float *host_floats = new float[N];

    float *f1_gpu;
    // MyStruct *struct_gpu;
    float *gpu_floats;

    hipMalloc((void**)&f1_gpu, N * sizeof(float));
    // hipMalloc((void**)&struct_gpu, sizeof(struct MyStruct));
    hipMalloc((void**)&gpu_floats, N * sizeof(float));

    f1_host[0] = 123;
    hipMemcpy(f1_gpu, f1_host, N * sizeof(float), hipMemcpyHostToDevice);

    host_floats[0] = 444;

    struct MyStruct mystruct;
    mystruct.somefloats = gpu_floats;
    hipMemcpy(gpu_floats, host_floats, N * sizeof(float), hipMemcpyHostToDevice);

    mykernel<<<dim3(1,1,1), dim3(32,1,1)>>>(mystruct, gpu_floats);

    hipMemcpy(host_floats, gpu_floats, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "host_floats[0]=" << host_floats[0] << std::endl;

    return 0;
}
