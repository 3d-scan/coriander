// I have/had a bug where internal function calls were processed to have struct arguments with pointers split out,
// just as for kernel calls
// This is a testcase to reproduce/fix/test that bug

#include "test_callinternal.h"

#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;

__device__ void somefunc(struct MyStruct mystruct) {
    mystruct.somefloats[0] += 3.0f;
}
