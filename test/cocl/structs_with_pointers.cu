
#include <hip/hip_runtime.h>
struct MyStruct {
    float *floats;
    int intvalue;
};

__device__ void foo_dev2(MyStruct *mystruct, float *data);

__device__ void foo_device(MyStruct *mystruct, float *data) {
    data[0] = mystruct[0].floats[0];
}

__global__ void foo(MyStruct *mystruct, float *data) {
    data[0] = mystruct[0].floats[0];
    foo_dev2(mystruct, data);
}

int main(int argc, char *argv[]) {
    return 0;
}
