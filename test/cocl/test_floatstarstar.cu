// double indirection, ie float **, in kernel parameter

// this test cuts all gpu buffers from one single gpu buffer

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct BoundedArray {
    float *bounded_array[8];
};

__global__ void run_bounded_array(struct BoundedArray boundedArray, int numBuffers, int N) {
    for(int i = 0; i < numBuffers; i++) {
        for(int j = 0; j < N; j++) {
            boundedArray.bounded_array[i][j] = 123.0f + i + 1 + j;
        }
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    const int numBuffers = 3;

    float *gpuArena;
    hipMalloc((void **)&gpuArena, numBuffers * N * 4 + 1024);

    struct BoundedArray boundedArray;
    float *hostFloats[numBuffers];

    for(int i = 0; i < numBuffers; i++) {
        boundedArray.bounded_array[i] = gpuArena + 256 + i * N;
        hostFloats[i] = new float[N];
    }

    run_bounded_array<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(boundedArray, numBuffers, N);

    for(int i = 0; i < numBuffers; i++) {
        hipMemcpy(hostFloats[i], boundedArray.bounded_array[i], N * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipStreamSynchronize(stream);

    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < 8; j++) {
            cout << hostFloats[i][j] << " ";
            assert(hostFloats[i][j] == 123.0f + 1 + i + j);
        }
        cout << endl;
    }

    for(int i=0; i < numBuffers; i++) {
        delete[] hostFloats[i];
    }
    hipFree(gpuArena);

    hipStreamDestroy(stream);

    return 0;
}
