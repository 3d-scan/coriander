// tests cuMemHostAlloc

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void incrValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostfloats;
    hipHostAlloc((void **)&hostfloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t devicefloats;
    hipMalloc(&devicefloats, N * sizeof(float));

    hostfloats[2] = 4.0f;
    hipMemcpyHtoDAsync(devicefloats, hostfloats, N * sizeof(float), stream);
    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)devicefloats, 2, 3.0f);
    hipMemcpyDtoHAsync(hostfloats, devicefloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);
    cout << "hostfloats[2] " << hostfloats[2] << endl;

    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)devicefloats, 2, 5.0f);
    hipMemcpyDtoHAsync(hostfloats, devicefloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);
    cout << "hostfloats[2] " << hostfloats[2] << endl;

    hipHostFree(hostfloats);
    hipFree(devicefloats);
    hipStreamDestroy(stream);

    return 0;
}
