// tests cuMemHostAlloc

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void incrValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    float *floats;
    hipHostAlloc((void **)&floats, N * sizeof(float), hipHostMallocPortable);

    floats[2] = 4.0f;
    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(floats, 2, 3.0f);
    cout << "floats[2] " << floats[2] << endl;

    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(floats, 2, 5.0f);
    cout << "floats[2] " << floats[2] << endl;

    hipHostFree(floats);

    return 0;
}
