/*
other things we should test:
- struct pointer, with offset
- multiple struct pointers, cut from same buffer
- getting values from various types of structs passed in
*/

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct MyStruct {
    float *p1;
    float *p2;
    float f1;
    float f2;
};

struct MyStruct2 {
    float *p1;
};

__global__ void struct_byvalue(struct MyStruct mystruct, float *out) {
    out[0] = mystruct.f1;
    out[1] = mystruct.f2;
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
}

void testbyvaluestruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    float *gpuFloats3;
    hipMalloc((void**)(&gpuFloats3), N * sizeof(float));

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    float *hostFloats1 = new float[N];
    float *hostFloats2 = new float[N];
    float *hostFloats3 = new float[N];
    float *hostOut = new float[N];

    struct MyStruct mystruct = {(float *)gpuFloats1, (float *)gpuFloats2, 3.0f, 8.0f};
    struct_byvalue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, (float *)gpuOut);

    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;
    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);
    assert(hostOut[0] == 3);
    assert(hostOut[1] == 8);

    hipFree(gpuFloats1);
    hipFree(gpuFloats2);
    hipFree(gpuFloats3);
    hipFree(gpuOut);

    delete[]hostFloats1;
    delete[]hostFloats2;
    delete[]hostFloats3;
    delete[]hostOut;

    hipStreamDestroy(stream);
}

struct GpusideStruct {
    float f1;
    float f2;
};

__global__ void struct_aspointer(struct GpusideStruct *mystruct, float *out) {
    out[0] = mystruct->f1;
    out[1] = mystruct->f2;
}

void testaspointerstruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostOut = new float[N];

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    struct GpusideStruct mystruct = { 5, 7 };
    struct GpusideStruct *gpu_mystruct;
    hipMalloc((void**)(&gpu_mystruct), sizeof(mystruct));
    hipMemcpy(gpu_mystruct, &mystruct, sizeof(mystruct), hipMemcpyHostToDevice);

    struct_aspointer<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(gpu_mystruct, gpuOut);

    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostOut[0] == 5);
    assert(hostOut[1] == 7);

    delete[]hostOut;

    hipStreamDestroy(stream);
}

// __global__ void kernel_twostructs(struct MyStruct *mystruct, struct MyStruct2 *mystruct2) {
//     mystruct->p1[0] = 9.0f;
//     mystruct->p2[0] = 10.0f;
//     mystruct2->p1[0] = 11.0f;
// }

// void testtwostructs() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     float *gpuFloats2;
//     cudaMalloc((void**)(&gpuFloats2), N * sizeof(float));

//     float *gpuFloats3;
//     cudaMalloc((void**)(&gpuFloats3), N * sizeof(float));

//     float *hostFloats1 = new float[N];
//     float *hostFloats2 = new float[N];
//     float *hostFloats3 = new float[N];

//     struct MyStruct mystruct = {(float *)gpuFloats1, (float *)gpuFloats2};
//     struct MyStruct2 mystruct2 = {(float *)gpuFloats3};

//     kernel_twostructs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(&mystruct, &mystruct2);

//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cudaMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cudaMemcpy(hostFloats3, gpuFloats3, 4 * sizeof(float), cudaMemcpyDeviceToHost);

//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats2[0] << endl;
//     cout << hostFloats3[0] << endl;

//     assert(hostFloats1[0] == 9);
//     assert(hostFloats2[0] == 10);
//     assert(hostFloats3[0] == 11);

//     cudaFree(gpuFloats1);
//     cudaFree(gpuFloats2);
//     cudaFree(gpuFloats3);

//     delete[]hostFloats1;
//     delete[]hostFloats2;
//     delete[]hostFloats3;

//     cuStreamDestroy(stream);
// }

// struct MyStructNoPtr {
//     float p1;
// };

// __global__ void kernel_structbyval_noptrs(struct MyStructNoPtr mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//         out[0] = mystruct1.p1;
//         out[1] = 5;
//     }
// }

// void teststructbyvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct MyStructNoPtr mystruct1 = {8.0f};

//     kernel_structbyval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);

//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);

//     delete[] hostFloats1;
//     cudaFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

// struct MyStruct2NoPtr {
//     float p1;
//     float p2;
// };

// __global__ void kernel_struct2byval_noptrs(struct MyStruct2NoPtr mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//     out[0] = mystruct1.p1;
//     out[1] = mystruct1.p2;
//     }
// }

// void teststruct2byvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1;
//     cuMemHostAlloc((void **)&hostFloats1, N * sizeof(float), CU_MEMHOSTALLOC_PORTABLE);

//     CUdeviceptr gpuFloats1;
//     cuMemAlloc(&gpuFloats1, N * sizeof(float));
//     cuMemcpyHtoDAsync((CUdeviceptr)(((float *)gpuFloats1)), hostFloats1, N * sizeof(float), stream);

//     struct MyStruct2NoPtr mystruct1 = {8.0f, 9.0f};

//     kernel_struct2byval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);
//     cuMemcpyDtoHAsync(hostFloats1, gpuFloats1, N * sizeof(float), stream);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);
//     assert(hostFloats1[1] == 9);

//     cuMemFreeHost(hostFloats1);
//     cuMemFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

// __global__ void kernel_twostructs_noptrs(struct MyStruct2NoPtr *mystruct, struct MyStructNoPtr *mystruct2, struct MyStructNoPtr mystruct3, float *out) {
//     if(threadIdx.x == 0) {
//     out[0] = mystruct->p1;
//     out[1] = mystruct->p2;
//     out[2] = mystruct2->p1;
//     out[3] = mystruct3.p1;
//     }
// }

// void testtwostructsNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct MyStruct2NoPtr mystruct = {5.0f, 6.0f};
//     struct MyStructNoPtr mystruct2 = {7.0f};
//     struct MyStructNoPtr mystruct3 = {8.0f};

//     kernel_twostructs_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(&mystruct, &mystruct2, mystruct3, (float *)gpuFloats1);
//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;
//     cout << hostFloats1[2] << endl;
//     cout << hostFloats1[3] << endl;

//     assert(hostFloats1[0] == 5);
//     assert(hostFloats1[1] == 6);
//     assert(hostFloats1[2] == 7);

//     cudaFree(gpuFloats1);

//     delete[] hostFloats1;

//     cuStreamDestroy(stream);
// }

int main(int argc, char *argv[]) {
    cout << "\ntestvaluestruct" << endl;
    testbyvaluestruct();

    cout << "\ntestaspointersstruct" << endl;
    testaspointerstruct();

    // cout << "\ntesttwostructs" << endl;
    // testtwostructs();

    // cout << "\teststruct2byvalNoPtr" << endl;
    // teststruct2byvalNoPtr();

    // cout << "\teststructbyvalNoPtr" << endl;
    // teststructbyvalNoPtr();

    // cout << "\ntesttwostructsNoPtr" << endl;
    // testtwostructsNoPtr();

    return 0;
}
