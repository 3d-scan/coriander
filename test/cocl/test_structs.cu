
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct MyStruct {
    float *p1;
    float *p2;
};

__global__ void struct_byvalue(struct MyStruct mystruct) {
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
}

__global__ void struct_aspointer(struct MyStruct *mystruct) {
    mystruct->p1[0] = 9.0f;
    mystruct->p2[0] = 10.0f;
}

void testbyvaluestruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats1)), hostFloats1, N * sizeof(float), stream);

    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats2, N * sizeof(float));
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats2)), hostFloats2, N * sizeof(float), stream);

    struct MyStruct mystruct = {(float *)deviceFloats1, (float *)deviceFloats2};
    struct_byvalue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct);

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats2);
    hipStreamDestroy(stream);
}

void testaspointerstruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats1)), hostFloats1, N * sizeof(float), stream);

    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats2, N * sizeof(float));
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats2)), hostFloats2, N * sizeof(float), stream);

    struct MyStruct mystruct = {(float *)deviceFloats1, (float *)deviceFloats2};
    struct_aspointer<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(&mystruct);

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats2);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    testbyvaluestruct();
    testaspointerstruct();
    return 0;
}
