
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>
#include "hipblas.h"

void transpose(float *in, float *out, int rows, int cols) {
    int newrows = cols;
    int newcols = rows;
    for(int oldrow = 0; oldrow < rows; oldrow++) {
        for(int oldcol = 0; oldcol < cols; oldcol++) {
            int newcol = oldrow;
            int newrow = oldcol;
            out[newrow * newcols + newcol] = in[oldrow * cols + oldcol];
        }
    }
}

void dump(float *M, int rows, int cols) {
    for(int row=0; row < rows; row++) {
        for(int col = 0; col < cols; col++) {
            cout << " " << M[row * cols + col];
        }
        cout << endl;
    }
}

void dumbMultiply(float *A, float *B, float *C, int M, int N, int K) {
    // assume row major
    for(int m=0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            float sum = 0;
            for(int k=0; k < K; k++) {
                sum += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = sum;
        }
    }
}

int main(int argc, char *argv[]) {
    const int M = 3;
    const int K = 2;
    const int N = 4;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    // float *hostA;
    // float *hostB;
    // float *hostC;
    // hipHostAlloc((void **)&hostA, M * K * sizeof(float), hipHostMallocPortable);
    // hipHostAlloc((void **)&hostB, K * N * sizeof(float), hipHostMallocPortable);
    // hipHostAlloc((void **)&hostC, M * N * sizeof(float), hipHostMallocPortable);

    float hostA[] = { 3, 5,
                      5, 8,
                      2, -1};
    float hostB[] = { 3, 5, 4, 1,
                      5, 8, 5, 7};
    float hostATrans[M * K];
    float hostBTrans[K * N];
    float hostC[M * N];
    float hostCTrans[M * N];

    transpose(hostA, hostATrans, M, K);
    transpose(hostB, hostBTrans, K, N);

    cout << "A:" << endl;
    dump(hostA, M, K);
    cout << "B:" << endl;
    dump(hostB, K, N);

    cout << "ATrans:" << endl;
    dump(hostATrans, K, M);
    cout << "BTrans:" << endl;
    dump(hostBTrans, N, K);

    hipDeviceptr_t deviceA;
    hipDeviceptr_t deviceB;
    hipDeviceptr_t deviceC;
    hipMalloc(&deviceA, M * K * sizeof(float));
    hipMalloc(&deviceB, K * N * sizeof(float));
    hipMalloc(&deviceC, M * N * sizeof(float));

    // hostFloats1[0] = 123.456f;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceA)),
        hostATrans,
        M * K * sizeof(float),
        stream
    );
    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceB)),
        hostBTrans,
        K * N * sizeof(float),
        stream
    );

    hipblasHandle_t blas;
    hipblasCreate(&blas);

    float alpha = 1;
    float beta = 0;
      hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
      &alpha,
      deviceA, M, deviceB, K, &beta, deviceC, M);

    hipblasDestroy(blas);

    // getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((float *)deviceFloats1);

    hipMemcpyDtoHAsync(hostCTrans, deviceC, M * N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << "C trans:" << endl;
    dump(hostCTrans, N, M);
    transpose(hostCTrans, hostC, N, M);
    cout << "C:" << endl;
    dump(hostC, M, N);

    float hostCCheck[M * N];
    dumbMultiply(hostA, hostB, hostCCheck, M, N, K);
    cout << "C check:" << endl;
    dump(hostCCheck, M, N);

    // for(int m=0; m < M; m++) {
    //     for(int n = 0; n < N; n++) {
    //         cout << " " << hostC[m * N + n];
    //     }
    //     cout << endl;
    // }

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    hipStreamDestroy(stream);

    return 0;
}
