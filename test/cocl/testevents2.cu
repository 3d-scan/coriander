// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 202400; // * 1024;

    float *hostfloats = new float[N];
    float *gpufloats;
    hipMalloc(&gpufloats, N * sizeof(float));

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1)>>>(gpufloats, N, 3.0f);
    cout << "queued kernel x" << endl;

    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    cout << "created event" << endl;

    hipEventRecord(event, stream);
    cout << "recoreded event" << endl;

    cout << "event finished? " << (hipEventQuery(event) == 0) << endl;

    hipEventSynchronize(event);
    cout << "synchronized event" << endl;

    cout << "event finished? " << (hipEventQuery(event) == 0) << endl;

    hipEventDestroy(event);
    cout << "destroyed event" << endl;

    // opencl:
    // clCreateUserEvent()   CL_EVENT_COMMAND_ EXECUTION_STATUS
    // clWaitForEvents(num_events, event_list);
    // clEnqueueMarkerWithWaitList
    // clGetEventInfo() 
    // clReleaseEvent

    // cuda:
    // cuEventCreate(CUEvent *, flags)
    // cuEventRecord(CUEvent, CUstream);  => puts into the stream
    // cuEventQuery(CUevent)
    // cuEventSynchronize(CUevent)
    // cuEventDestroy

    cout << "finished" << endl;

    hipStreamDestroy(stream);
    hipEventDestroy(event);
    hipFree(gpufloats);

    return 0;
}
