// What if we are given a device pointer that is offset from any of the device pointers we provided to the client?
//
// This file is a test-case for this.  Then we can look at handling that...

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

// __global__ void getValue(float *data, int idx, float value) {
//     if(threadIdx.x == 0) {
//         data[idx] += value;
//     }
// }

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostfloats;
    hipHostAlloc((void **)&hostfloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t devicefloats;
    hipMalloc(&devicefloats, N * sizeof(float));

    hostfloats[128] = 123.456f;
    hostfloats[129] = 444.0f;
    hostfloats[130] = 321.0f;
    hostfloats[131] = 111.0f;

    // now we will copy 16 bytes, starting at location 128...
    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)devicefloats) + 64),
        hostfloats + 128,
        4 * sizeof(float),
        stream
    );
    hipStreamSynchronize(stream);
    // now copy back entire buffer
    hostfloats[64] = 0.0f;
    hostfloats[65] = 0.0f;
    hostfloats[66] = 0.0f;
    hostfloats[67] = 0.0f;
    hipMemcpyDtoHAsync(hostfloats, devicefloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostfloats[64] << endl;
    cout << hostfloats[65] << endl;
    cout << hostfloats[66] << endl;
    cout << hostfloats[67] << endl;

    hipHostFree(hostfloats);
    hipFree(devicefloats);
    hipStreamDestroy(stream);

    return 0;
}
