// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 102400;

    hipStream_t newstream;
    hipStreamCreateWithFlags(&newstream, 0);
    cout << "got stream" << endl;

    float *floats;
    hipHostAlloc((void **)&floats, N * sizeof(float), hipHostMallocPortable);

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1)>>>(floats, N, 3.0f);
    cout << "queued kernel x" << endl;

    hipCtxSynchronize();
    cout << "finished" << endl;

    hipHostFree(floats);

    hipStreamDestroy(newstream);

    return 0;
}
