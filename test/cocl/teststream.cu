// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 102400;

    hipStream_t newstream;
    hipStreamCreateWithFlags(&newstream, 0);
    cout << "got stream" << endl;

    float *floats;
    hipHostAlloc((void **)&floats, N * sizeof(float), hipHostMallocPortable);

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1), 0, newstream>>>(floats, N, 3.0f);
    cout << "queued kernel x" << endl;

    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);

    // cuStreamWaitEvent(newstream, event, 0);

    hipCtxSynchronize();
    cout << "finished" << endl;

    hipHostFree(floats);

    hipStreamDestroy(newstream);

    return 0;
}
