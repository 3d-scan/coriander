// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

// extern "C" {
//     void configureKernel(const char *, const char *);
//     void _Z17setKernelArgInt64l(long long);
//     // size_t clConfigureCall(unsigned long, unsigned int, unsigned long, unsigned int, size_t shared=0, void *stream=0);
// // size_t cudaConfigureCall(
// //         dim3 a,
// //         dim3 b, size_t sharedmem=0, void *stream=0);
// }

int main(int argc, char *argv[]) {
    int N = 1024;

    float *gpuFloats;
    hipMalloc((void**)(&gpuFloats), N * sizeof(float));

    // cout << "1" << endl;
    // configureKernel("foo", "kernel void foo(long arg){}");
    // _Z17setKernelArgInt64l(123);
    // clConfigureCall(1, 3, 5, 6, 0, 0);
    // cudaConfigureCall(dim3(0, 1, 2), dim3(5, 2, 3), 0, 0);
    // cout << "2" << endl;
    // configureKernel("foo", "kernel void foo(long arg){}");

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 123.0f);

    float hostFloats[4];
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 222.0f);
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;

    hostFloats[2] = 444.0f;
    hipMemcpy(gpuFloats, hostFloats, 4 * sizeof(float), hipMemcpyHostToDevice);
    hostFloats[2] = 555.0f;
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;

    hipFree(gpuFloats);

    return 0;
}
