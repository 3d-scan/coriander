// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

void hostside_opencl_funcs_init();

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    hostside_opencl_funcs_init();

    int N = 1024;

    float *gpuFloats;
    hipMalloc((void**)(&gpuFloats), N * sizeof(float));
    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 123.0f);

    float hostFloats[4];
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 222.0f);
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "hostFloats[2] " << hostFloats[2] << endl;

    hipFree(gpuFloats);

    return 0;
}
