// This is from Eigen unsupported/test folder
// will gradually uncomment lines, and fix the issues which thus arise

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
// #define EIGEN_TEST_FUNC cxx11_tensor_cuda
#define EIGEN_USE_GPU

// #if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
// #include <hip/hip_fp16.h>
// #endif
#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

using Eigen::Tensor;

// #include "main.h"

void test_cuda_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> in2(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> out(Eigen::array<Eigen::DenseIndex, 1>(2));
  // in1.setRandom();
  // in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  // hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  // hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);

  // Eigen::CudaStreamDevice stream;
  // Eigen::GpuDevice gpu_device(&stream);

  // Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
  //     d_in1, Eigen::array<Eigen::DenseIndex, 1>(2));
  // Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
  //     d_in2, Eigen::array<Eigen::DenseIndex, 1>(2));
  // Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
  //     d_out, Eigen::array<Eigen::DenseIndex, 1>(2));

  // gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  // assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
  //                        gpu_device.stream()) == hipSuccess);
  // assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  // for (int i = 0; i < 2; ++i) {
  //   VERIFY_IS_APPROX(
  //       out(Eigen::array<Eigen::DenseIndex, 1>(i)),
  //       in1(Eigen::array<Eigen::DenseIndex, 1>(i)) + in2(Eigen::array<Eigen::DenseIndex, 1>(i)));
  // }

  cout << "d_out " << d_out << endl;
  cout << "d_in2 " << d_in2 << endl;

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}

// int main(int argc, char *argv[]) {
//   test_cuda_elementwise_small();
//   return 0;
// }
