
#include <hip/hip_runtime.h>
struct MyStruct {
    float floatvalue;
    int intvalue;
};

__device__ __host__ float sumStruct(struct MyStruct **p_structs, int N) {
    float sum = 0;
    for(int i = 0; i < N; i++) {
        struct MyStruct *mystruct = p_structs[i];
        sum += mystruct->floatvalue + float(mystruct->intvalue) * 3.5f;
    }
    return sum;
}

__global__ void mykernel(float *data, MyStruct **p_structs, int N) {
    data[0] = sumStruct(p_structs, N);
}
