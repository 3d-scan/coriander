
#include <hip/hip_runtime.h>
namespace mynamespace {
    namespace subnamespace {
        __device__ __host__ class Foo {
        public:
            int a;
            Foo() {

            }
            ~Foo() {

            }
            void somefunc() {

            }
        };
    }

    __device__ __host__ class Bar : public subnamespace::Foo {
    public:
        Foo foo;
        int b;
        Bar() {

        }
        ~Bar() {

        }
        void somefunc2() {

        }
    };
}

__global__ void doSomething(mynamespace::Bar bar) {
    bar.a = 123;
}
