// tests cuMemHostAlloc

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

// __global__ void setValue(float *data, int idx, float value) {
//     if(threadIdx.x == 0) {
//         data[idx] = value;
//     }
// }

__global__ void incrValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    float *floats;
    hipHostAlloc((void **)&floats, N * sizeof(float), hipHostMallocPortable);
    // cudaMalloc((void**)(&gpuFloats), N * sizeof(float));

    floats[2] = 4.0f;
    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(floats, 2, 3.0f);
    cout << "floats[2] " << floats[2] << endl;

    // float hostFloats[4];
    // cudaMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), cudaMemcpyDeviceToHost);

    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(floats, 2, 5.0f);
    // cudaMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), cudaMemcpyDeviceToHost);
    cout << "floats[2] " << floats[2] << endl;

    // hostFloats[2] = 444.0f;
    // cudaMemcpy(gpuFloats, hostFloats, 4 * sizeof(float), cudaMemcpyHostToDevice);
    // hostFloats[2] = 555.0f;
    // cudaMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), cudaMemcpyDeviceToHost);
    cout << "floats[2] " << floats[2] << endl;

    // cudaFree(gpuFloats);
    hipHostFree(floats);

    return 0;
}
