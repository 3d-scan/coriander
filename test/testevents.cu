// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 128; // * 1024;

    float *floats;
    hipHostAlloc((void **)&floats, N * sizeof(float), hipHostMallocPortable);

    // floats[2] = 4.0f;
    longKernel<<<dim3(1, 1, 1), dim3(32, 1, 1)>>>(floats, N, 3.0f);
    cout << "queued kernel x" << endl;

    // cuCtxSynchronize();
    cout << "finished" << endl;

    hipHostFree(floats);

    return 0;
}
